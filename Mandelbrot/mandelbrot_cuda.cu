#include "hip/hip_runtime.h"

/* 
Author: Jackson Crandell
Class: ECE 4122
Last Date Modified: 12/07/21 
 
Description: Mandelbrot set with CUDA optimization.

Reference: http://selkie.macalester.edu/csinparallel/modules/CUDAArchitecture/build/html/1-Mandelbrot/Mandelbrot.html
*/

//nvcc mandelbrot_cuda.cu -lglut -lGL -lGLU -lGLEW -lgomp

#include <fstream>
#include <iostream>
#include <stdio.h>
#include <GL/glut.h>
#include <sstream>
#include <string>
#include <vector>
#include <complex>    
#include <iomanip>
#include <omp.h>

GLuint window;

// Must be square window
// Preferable a window that is a multiple of 32, otherwise it may not work
static int dim = 512;
GLuint width = dim, height = dim;
static int n = 512;
static int m = 512;
static int max_iter = 1000;
double frame_x = -0.5;
double frame_y = 0;
double scale = 3;

int lastx = 0;
int lasty = 0;

void display();
void reshape(int width, int height);
void draw_point(int i, int j, int k);
void special(int key, int x, int y);
void mouse(int button, int state, int x, int y);



/**
 * Cuda kernel function that calculates if a number is in the Mandelbrot set. 
 * This is defined by as any complex number, c, such that z = z^2 + c remains bounded.
 * For our purpose, we find if the magnitude of the number is < 2 then it is in the set. 
 * More iterations results in higher accuracy.
 *
 * @param n - holds number of points to calculate
 * @param h - holds heat calculates
 * @param g - temp array to hold heat calculations
 * 
 */
__device__ uint32_t mandelbrot_set(double cr, double ci, int max_iter) {
    double real = 0;
    double imag = 0;
    double realsqr = 0;
    double imagsqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++){
		imag = real * imag;
		imag += imag;
		imag += ci;
		real = realsqr - imagsqr + cr;
		realsqr = real * real;
		imagsqr = imag * imag;
		
		if (realsqr + imagsqr > 4.0) break;
    }
	
    return i;
}

/**
 * Cuda kernel function that turns each (x,y) coordinate into a complex number.
 * The it runs the mandel_double kernel.
 *
 * @param counts holds pixel values
 * @param xmin minimum x value of screen
 * @param ymin minimum y value of screen
 * @param step size to move left and right by. Needed when zooming in
 * @param max_iter number of iterations to run until. (Higher iterations lead to higher accuracy)
 * @param dim dimension of window
 * 
 */
__global__ void mandel_kernel(uint32_t *counts, double xmin, double ymin, double step, int max_iter, int dim) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++){
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = mandelbrot_set(cr, ci, max_iter);
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim && tId < (dim * dim) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = mandelbrot_set(cr, ci, max_iter);
    }
    
}

/**
 * Cuda kernel function that turns each (x,y) coordinate into a complex number.
 * The it runs the mandel_double kernel.
 *
 * @param frame_x controls where to render fractal in x -changed via panning
 * @param frame_y controls where to render fractal in y-changed via panning
 * @param scale scaling factor of fractal 
 * @param dev_counts 
 * 
 */
void display_mandelbrot(double frame_x, double frame_y, double scale, uint32_t *dev_counts)
{
    double start = omp_get_wtime();
    dim3 numBlocks(dim,dim);
    double xmin = frame_x - (scale/2);
    double ymin = frame_y - (scale/2);
    double step = scale / dim;
    hipError_t err = hipSuccess;
    mandel_kernel<<<n, m>>>(dev_counts, xmin , ymin, step, max_iter, dim);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel!\n");
        exit(EXIT_FAILURE);
    }
	int *data = (int *)malloc(dim * dim * sizeof(uint32_t));
    err = hipMemcpy(data, dev_counts, dim * dim * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy dev_counts back\n");
        exit(EXIT_FAILURE);
    }
    double stop = omp_get_wtime();
    //printf("Blocks: %d\tThreads per Block: %d\tSize:%dx%d\tDepth: %d\tTime: %f\n",n, m, dim, dim, max_iter, stop - start);

    for (int i = 0; i < width; ++i)
    {
        for (int j = 0; j < height; ++j)
		{
			draw_point(j, i, data[i* dim + j]);
		} 
    }
}

/**
 * Assigns a color based returned iteration.
 *
 * @param color represents the iteration returned.
 * 
 */
void set_color(int color)
{
    // Set color to black if number is in max iter
	if (color == max_iter)
	{
		glColor3f(0, 0, 0);
	}
	else
	{
        float freq = 6.3 / max_iter;
        glColor3f((sin(freq * color + 5)),sin(freq * color + 3),sin(freq * color + 1));
	}
}

/**
 * Draw the point on the screen.
 * 
 */
void draw_point(int i, int j, int color) {
	glBegin(GL_POINTS);
	set_color(color);
	glVertex2i(i, height - j);
	glEnd();
}

/**
 * Handles mouse input. Use the scroll wheel to zoom in and out
 * of the scene
 * 
 * @param button holds the different mouse inputs (i.e. scroll wheel in our case)
 * @param x holds mouse x location when key is pushed
 * @param y holds mouse y location when key is pushed
 * 
 */
void mouse(int button, int state, int x, int y) {
	// Wheel reports as button 3(scroll up) and button 4(scroll down)
   if ((button == 3) || (button == 4)) // It's a wheel event
   {
       // Each wheel event reports like a button click, GLUT_DOWN then GLUT_UP
       if (state == GLUT_UP) return; // Disregard redundant GLUT_UP events
	   if (button == 3)
	   {
			scale *= 0.80;
	   }
	   else 
	   {
			scale *= 1.25;
	   }
   }

    // Panning
    lastx = x;
    lasty = y;
    display();
}

/**
 * Handles keyboard input. Use the arrow keys to control the camera to 
 * simulate the tetrahedron moving.
 * 
 * @param key holds value of key pushed
 * @param x holds mouse x location when key is pushed
 * @param y holds mouse y location when key is pushed
 * 
 */
void special(int key, int x, int y) {
	switch (key) {
		case GLUT_KEY_UP:
			frame_y -= 20 * scale / dim;
			break;
		case GLUT_KEY_DOWN:
			frame_y += 20 * scale / dim;
			break;
		case GLUT_KEY_RIGHT:
			frame_x += 20 * scale / dim;
			break;
		case GLUT_KEY_LEFT:
			frame_x -= 20 * scale / dim;
			break;
	}
}

/**
 * Displays Mandelbrot set.
 */
void display()
{
	glViewport(0, 0, width, height);
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, width, 0, height);
	glMatrixMode(GL_MODELVIEW);	
	glLoadIdentity();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glPointSize(1.0);

    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_mandelbrot(frame_x, frame_y, scale, dev_counts);

	glutSwapBuffers();
	glutPostRedisplay();
}

void reshape(int w, int h)
{
	width = w; height = h;
	glViewport(0, 0, width, height);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, width, 0, height);
	glMatrixMode(GL_MODELVIEW);	
	glLoadIdentity();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glPointSize(1.0);

    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_mandelbrot(frame_x, frame_y, scale, dev_counts);

}

/**
 * Handles mouse input to allow for panning.
 * 
 * @param key holds value of key pushed
 * @param x holds mouse x location when key is pushed
 * @param y holds mouse y location when key is pushed
 * 
 */
void mouseMovement(int x, int y)
{
    int diffx = x - lastx; 
    int diffy = y - lasty; 
    lastx = x;
    lasty = y;
    diffx = (diffx > 1.0f) ? 1.0f : diffx;
    diffx = (diffx < -1.0f) ? -1.0f : diffx;
    diffy = (diffy > 1.0f) ? 1.0f : diffy;
    diffy = (diffy < -1.0f) ? -1.0f : diffy;
	frame_y -= diffy * 10 * scale / dim;
	frame_x -= diffx * 10 * scale / dim;
}



int main(int argc, char** argv)
{
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(width, height);
	glutInitWindowPosition(100, 100);
	glutInit(&argc, argv);

	window = glutCreateWindow("OpenGL Mandelbrot Fractal Set");
	glutReshapeFunc(reshape);
	glutDisplayFunc(display);
    glutSpecialFunc(special);
    glutMouseFunc(mouse);
    glutMotionFunc(mouseMovement);
    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_mandelbrot(frame_x, frame_y, scale, dev_counts);
	glutMainLoop();
    hipFree(dev_counts);
	return 0;
}
