#include "hip/hip_runtime.h"
//nvcc mandelbrot_cuda.cu -lglut -lGL -lGLU -lGLEW -lgomp && time ./a.out

#include <fstream>
#include <iostream>
#include <stdio.h>
#include <GL/glut.h>
#include <sstream>
#include <string>
#include <vector>
#include <complex>    
#include <iomanip>
#include <omp.h>

GLuint window;

static int dim = 512;
GLuint width = dim, height = dim;
static int n = 512;
static int m = 512;
static int max_iter = 10000;
double xcen = -0.5;
double ycen = 0;
double scale = 3;

void display();
void reshape(int width, int height);
void draw_point(int i, int j, int k);
void special(int key, int x, int y);
void mouse(int button, int state, int x, int y);
//void idle();

/* the mandelbrot set is defined as all complex numbers c such that the 
   equation z = z^2 + c remains bounded. In practice, we calculate max_iter
   iterations of this formula and if the magnitude of z is < 2 we assume it
   is in the set. The greater max_iters the more accurate our representation */
__device__ uint32_t mandel_double(double cr, double ci, int max_iter) {
    double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++){
		zi = zr * zi;
		zi += zi;
		zi += ci;
		zr = zrsqr - zisqr + cr;
		zrsqr = zr * zr;
		zisqr = zi * zi;
		
    //the fewer iterations it takes to diverge, the farther from the set
		if (zrsqr + zisqr > 4.0) break;
    }
	
    return i;
}

/* turn each x y coordinate into a complex number and run the mandelbrot formula on it */
__global__ void mandel_kernel(uint32_t *counts, double xmin, double ymin, double step, int max_iter, int dim) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++){
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = mandel_double(cr, ci, max_iter);
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim && tId < (dim * dim) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = mandel_double(cr, ci, max_iter);
    }
    
}

void display_double(double xcen, double ycen, double scale, uint32_t *dev_counts)
{
    double start = omp_get_wtime();
    dim3 numBlocks(dim,dim);
    double xmin = xcen - (scale/2);
    double ymin = ycen - (scale/2);
    double step = scale / dim;
    hipError_t err = hipSuccess;
    mandel_kernel<<<n, m>>>(dev_counts, xmin , ymin, step, max_iter, dim);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel!\n");
        exit(EXIT_FAILURE);
    }
	int *data = (int *)malloc(dim * dim * sizeof(uint32_t));
    err = hipMemcpy(data, dev_counts, dim * dim * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy dev_counts back\n");
        exit(EXIT_FAILURE);
    }
    double stop = omp_get_wtime();
    //printf("Blocks: %d\tThreads per Block: %d\tSize:%dx%d\tDepth: %d\tTime: %f\n",n, m, dim, dim, max_iter, stop - start);

    for (int i = 0; i < width; ++i)
    {
        for (int j = 0; j < height; ++j)
		{
			draw_point(j, i, data[i* dim + j]);
		} 
    }
}


int main(int argc, char** argv)
{
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(width, height);
	glutInitWindowPosition(100, 100);
	glutInit(&argc, argv);

	window = glutCreateWindow("OpenGL Mandelbrot Fractal Set");
	glutReshapeFunc(reshape);
	glutDisplayFunc(display);
    glutSpecialFunc(special);
    glutMouseFunc(mouse);
    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_double(xcen, ycen, scale, dev_counts);
	glutMainLoop();
    hipFree(dev_counts);
	return 0;
}

void set_color(int color)
{
	if (color == max_iter)
	{
		glColor3f(0, 0, 0);
	}
	else
	{
        // TODO: Set different colors
        float freq = 6.3 / max_iter;
        glColor3f((sin(freq * color + 3)),sin(freq * color + 5),sin(freq * color + 1));
		//glColor3f((double)color / 100.0, (double)color / 100.0, (double)color / 100.0);
	}
}


void draw_point(int i, int j, int color) {
	glBegin(GL_POINTS);
	set_color(color);
	glVertex2i(i, height - j);
	glEnd();
}

void mouse(int button, int state, int x, int y) {
	// Wheel reports as button 3(scroll up) and button 4(scroll down)
   if ((button == 3) || (button == 4)) // It's a wheel event
   {
       // Each wheel event reports like a button click, GLUT_DOWN then GLUT_UP
       if (state == GLUT_UP) return; // Disregard redundant GLUT_UP events
	   if (button == 3)
	   {
			scale *= 0.80;
	   }
	   else 
	   {
			scale *= 1.25;
	   }
   }
   display();
}


void special(int key, int x, int y) {
	switch (key) {
		case GLUT_KEY_UP:
			ycen -= 20 * scale / dim;
			break;
		case GLUT_KEY_DOWN:
			ycen += 20 * scale / dim;
			break;
		case GLUT_KEY_RIGHT:
			xcen += 20 * scale / dim;
			break;
		case GLUT_KEY_LEFT:
			xcen -= 20 * scale / dim;
			break;
	}
}

void display()
{
	glViewport(0, 0, width, height);
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, width, 0, height);
	glMatrixMode(GL_MODELVIEW);	
	glLoadIdentity();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glPointSize(1.0);

    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_double(xcen, ycen, scale, dev_counts);

	glutSwapBuffers();
	glutPostRedisplay();
}

/* void idle()
{
    glutPostRedisplay();
} */

void reshape(int w, int h)
{
	width = w; height = h;
	glViewport(0, 0, width, height);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, width, 0, height);
	glMatrixMode(GL_MODELVIEW);	
	glLoadIdentity();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glPointSize(1.0);

    hipError_t err = hipSuccess;

    uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate dev_counts\n");
        exit(EXIT_FAILURE);
    }
    display_double(xcen, ycen, scale, dev_counts);

}

